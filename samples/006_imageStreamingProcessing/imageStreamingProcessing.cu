#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include "../utils/utils.hpp"

__global__ void grayscaleKernel(hipTextureObject_t texObj, unsigned char* out, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    
    uchar4 pixel = tex2D<uchar4>(texObj, x, y);
    out[y * width + x] = (pixel.x + pixel.y + pixel.z) / 3;
}

void processSingleImage(ImageStocker& stocker, int index, hipStream_t stream)
{

}

class CudaImageResource {
    public: 
        int width;
        int height;
        hipArray* cuArray = nullptr;
        hipTextureObject_t texObj = 0;
        unsigned char* d_output = nullptr;
        hipStream_t stream = nullptr;
        cv::Mat source;
        cv::Mat output;
        dim3 block;
        dim3 grid;
        void Destroy() {
            if (texObj != 0) hipDestroyTextureObject(texObj);
            if (cuArray != nullptr) hipFreeArray(cuArray);
            if (d_output != nullptr) hipFree(d_output);
            if (stream != nullptr) hipStreamDestroy(stream);
        }
};

void use_StreamProcess(ImageStocker stocker)
{
    // Create Stream
    std::vector<CudaImageResource> resources;
    int N = stocker.NumImages();
    resources.resize(N);
    for (int i = 0; i < N; i++)
    {
        hipStream_t stream;
        hipStreamCreate(&stream);
        resources[i].stream = stream;
    }

    // Initialize
    for (int i = 0; i < N; i++)
    {
        // hipArray
        cv::cvtColor(stocker.Get(i), resources[i].source, cv::COLOR_BGR2BGRA);
        resources[i].width = resources[i].source.cols;
        resources[i].height = resources[i].source.rows;
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
        hipMallocArray(&resources[i].cuArray, &channelDesc, resources[i].width, resources[i].height);

        // Texture Object
        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = resources[i].cuArray;
        hipTextureDesc texDesc = {};
        texDesc.addressMode[0] = hipAddressModeClamp;
        texDesc.addressMode[1] = hipAddressModeClamp;
        texDesc.filterMode = hipFilterModePoint;
        texDesc.readMode = hipReadModeElementType;
        texDesc.normalizedCoords = 0;
        hipCreateTextureObject(&resources[i].texObj, &resDesc, &texDesc, nullptr);

        hipMalloc(&resources[i].d_output, resources[i].width * resources[i].height * sizeof(unsigned char));

        // Kernel call
        resources[i].block = dim3(4, 4);
        resources[i].grid = dim3((resources[i].width + 3) / 4, (resources[i].height + 3) / 4);

        // Save Image
        resources[i].output = cv::Mat(resources[i].height, resources[i].width, CV_8UC1);
    }

    // Set Tasks
    for (int i = 0; i < N; i++)
    {
        hipMemcpy2DToArrayAsync(resources[i].cuArray, // target
            0, 0, resources[i].source.ptr<uchar4>(), resources[i].source.step, resources[i].width * sizeof(uchar4), resources[i].height, // image 
            hipMemcpyHostToDevice, resources[i].stream); // direction
    }

        for (int i = 0; i < N; i++)
    {
        grayscaleKernel<<<resources[i].grid, resources[i].block, 0, resources[i].stream>>>(resources[i].texObj, resources[i].d_output, resources[i].width, resources[i].height);
    }
        for (int i = 0; i < N; i++)
    {
        hipMemcpyAsync(resources[i].output.data, resources[i].d_output, resources[i].width * resources[i].height * sizeof(unsigned char), hipMemcpyDeviceToHost, resources[i].stream);
    }

    for (int i = 0; i < N; i++)
    {
        hipStreamSynchronize(resources[i].stream);
    }

    for (int i = 0; i < N; i++)
    {
        std::string outPath = "006_cuda-stream_img_" + ZeroPadding(2, i) + ".png";
        if (!cv::imwrite(outPath, resources[i].output))
        {
            std::cerr << "Failed to save iamge to " << outPath << std::endl;
            return ;
        }
        std::cout << "Saved image to: " << outPath << std::endl;
        resources[i].Destroy();
    }
}

int main(int argc, char** argv)
{
    if (argc < 2)
    {
        std::cerr << "Usage: " << argv[0] << " <image_dir>" << std::endl;
        return 1;
    }

    std::string imgDir = argv[1];
    ImageStocker stocker;
    stocker.LoadImages(imgDir);
    use_StreamProcess(stocker);
    return -1;
}