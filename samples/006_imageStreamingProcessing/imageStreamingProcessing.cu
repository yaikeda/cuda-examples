#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include "../utils/utils.hpp"

__global__ void grayscaleKernel(hipTextureObject_t texObj, unsigned char* out, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    
    uchar4 pixel = tex2D<uchar4>(texObj, x, y);
    out[y * width + x] = (pixel.x + pixel.y + pixel.z) / 3;
}

void processSingleImage(ImageStocker& stocker, int index)
{
    // hipArray
    cv::Mat imageBGRA;
    cv::cvtColor(stocker.Get(index), imageBGRA, cv::COLOR_BGR2BGRA);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
    hipArray* cuArray;
    hipMallocArray(&cuArray, &channelDesc, imageBGRA.cols, imageBGRA.rows);
    hipMemcpy2DToArray(cuArray, // target
        0, 0, imageBGRA.ptr<char4>(), imageBGRA.step, imageBGRA.cols * sizeof(uchar4), imageBGRA.rows, // image 
        hipMemcpyHostToDevice); // direction

    // Texture Object
    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;
    hipTextureDesc texDesc = {};
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);

    int width = imageBGRA.cols;
    int height = imageBGRA.rows;
    unsigned char* d_output;
    hipMalloc(&d_output, width * height * sizeof(unsigned char));

    // Kernel call
    dim3 block(16, 16);
    dim3 grid((width + 15) / 16, (height + 15) / 16);
    grayscaleKernel<<<grid, block>>>(texObj, d_output, width, height);

    // Save Image
    cv::Mat outputGray(height, width, CV_8UC1);
    hipMemcpy(outputGray.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);
    std::string outPath = "006_cuda-stream_img_" + ZeroPadding(2, index) + ".png";
    if (!cv::imwrite(outPath, outputGray))
    {
        std::cerr << "Failed to save iamge to " << outPath << std::endl;
        return ;
    }
    std::cout << "Saved image to: " << outPath << std::endl;

    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
    hipFree(d_output);
}

void use_StreamProcess(ImageStocker stocker)
{
    // Create Stream
    std::vector<hipStream_t> streams;
    int N = stocker.NumImages();
    streams.resize(N);
    for (int i = 0; i < N; i++)
    {
        hipStreamCreate(&streams[i]);
        processSingleImage(stocker, i);
    }
}

int main(int argc, char** argv)
{
    if (argc < 2)
    {
        std::cerr << "Usage: " << argv[0] << " <image_dir>" << std::endl;
        return 1;
    }

    std::string imgDir = argv[1];
    ImageStocker stocker;
    stocker.LoadImages(imgDir);
    use_StreamProcess(stocker);
    return -1;
}