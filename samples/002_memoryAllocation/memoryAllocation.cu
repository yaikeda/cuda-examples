#include <hip/hip_runtime.h>
#include <iostream>

const size_t SIZE = 1L << 30;

bool isSuccess(hipError_t err) {
    if (err != hipSuccess)
    {
        std::cerr << "cuda memory allocation failed: " << hipGetErrorString(err) << std::endl;
        return false;
    }
    return true;
}

void* use_cudaMalloc() {
    void* device_ptr = nullptr;
    
    if (isSuccess(hipMalloc(&device_ptr, SIZE)))
    {
        std::cout << "hipMalloc succeeded. Pointer = " << device_ptr << std::endl;
        return device_ptr;
    }
    return nullptr;
}

void* use_cudaMallocManaged() {
    void* unified_ptr = nullptr;
    
    if (isSuccess(hipMallocManaged(&unified_ptr, SIZE)))
    {
        std::cout << "hipMallocManaged succeeded. Pointer = " << unified_ptr << std::endl;
        return unified_ptr;
    }
    return nullptr;
}

void* use_cudaHostAlloc() {
    void* host_ptr = nullptr;
    if(isSuccess(hipHostAlloc(&host_ptr, SIZE, hipHostMallocDefault)))
    {
        std::cout << "hipHostAlloc succeeded. Pointer = " << host_ptr << std::endl;
        return host_ptr;
    }
    return nullptr;
}

int main() 
{
    void* device_ptr = use_cudaMalloc();
    void* unified_ptr = use_cudaMallocManaged();
    void* host_ptr = use_cudaHostAlloc();

    std::cin.get(); // pause

    hipFree(device_ptr);
    hipFree(unified_ptr);
    hipFree(host_ptr);
}