#include <hip/hip_runtime.h>
#include <iostream>

const size_t SIZE = 1L << 30;

bool isSuccess(hipError_t err) {
    if (err != hipSuccess)
    {
        std::cerr << "cuda memory allocation failed: " << hipGetErrorString(err) << std::endl;
        return false;
    }
    return true;
}

void* use_cudaMalloc() {
    void* device_ptr = nullptr;
    
    if (isSuccess(hipMalloc(&device_ptr, SIZE)))
    {
        std::cout << "hipMalloc succeeded. Pointer = " << device_ptr << std::endl;
        return device_ptr;
    }
    return nullptr;
}

void* use_cudaMallocManaged() {
    void* unified_ptr = nullptr;
    
    if (isSuccess(hipMallocManaged(&unified_ptr, SIZE)))
    {
        std::cout << "hipMallocManaged succeeded. Pointer = " << unified_ptr << std::endl;
        return unified_ptr;
    }
    return nullptr;
}

void* use_cudaHostAlloc() {
    void* host_ptr = nullptr;
    if(isSuccess(hipHostAlloc(&host_ptr, SIZE, hipHostMallocDefault)))
    {
        std::cout << "hipHostAlloc succeeded. Pointer = " << host_ptr << std::endl;
        return host_ptr;
    }
    return nullptr;
}

void* use_cudaHostAlloc_cudaHostAllocMapped()
{
    void* h_ptr = nullptr;

    if(isSuccess(hipHostAlloc(&h_ptr, SIZE, hipHostMallocMapped)))
    {
        void* d_ptr = nullptr;
        if(isSuccess(hipHostGetDevicePointer(&d_ptr, h_ptr, 0))) // flag must be zero for now
        {
            std::cout << "Zero-copy mapping succeeded. Host = " << h_ptr << ", Device =" << d_ptr << std::endl;
            return h_ptr;
        }
        hipFree(h_ptr);
        return nullptr;
    }
    return nullptr;
}

int main() 
{
    void* device_ptr = use_cudaMalloc();
    void* unified_ptr = use_cudaMallocManaged();
    void* host_ptr = use_cudaHostAlloc();
    void* host_ptr2 = use_cudaHostAlloc_cudaHostAllocMapped();

    std::cin.get(); // pause

    hipFree(device_ptr);
    hipFree(unified_ptr);
    hipFree(host_ptr);
    hipFree(host_ptr2);
}