#include <hip/hip_runtime.h>
#include <iostream>

const size_t SIZE = 1L << 30;

bool isSuccess(hipError_t err) {
    if (err != hipSuccess)
    {
        std::cerr << "hipMallocManaged failed: " << hipGetErrorString(err) << std::endl;
        return false;
    }
    return true;
}

void* use_cudaMalloc() {
    void* device_ptr = nullptr;
    size_t size = SIZE; 

    if (isSuccess(hipMalloc(&device_ptr, size)))
    {
        std::cout << "hipMalloc succeeded. Pointer = " << device_ptr << std::endl;
        return device_ptr;
    }
    return nullptr;
}

void* use_cudaMallocManaged() {
    void* unified_ptr = nullptr;
    size_t size = SIZE;

    if (isSuccess(hipMallocManaged(&unified_ptr, size)))
    {
        std::cout << "hipMallocManaged succeeded. Pointer = " << unified_ptr << std::endl;
        return unified_ptr;
    }
    return nullptr;
}

int main() 
{
    void* device_ptr = use_cudaMalloc();
    void* unified_ptr = use_cudaMallocManaged();
    
    std::cin.get(); // pause

    hipFree(device_ptr);
    hipFree(unified_ptr);
}